#include "hip/hip_runtime.h"
// This is a parallel Breath First Search (BFS) Implementation using CUDA
// Input: randomly generated graph in form of adjacent matrix stored in 'mygraph.txt'
// BFS starts from node 0 and stops when the logical BFS tree is formed
// usage: ./<out> <graph_size> <mygraph.txt

#include <cstdlib>
#include <iostream>
#include <time.h>

using namespace std;

__global__ void kernel(bool* adj_mat, const int N, bool* visited, int* frontier, bool* new_frontier, bool* par_mat, int* cap_mat, int* cap_max_mat){
	int row_idx = frontier[blockIdx.x+1];
	long offset = N * row_idx;

	// update new_frontier in threads
	int col_idx = threadIdx.x;
	long offset2 = N * col_idx;
	if(adj_mat[offset + col_idx] && (cap_mat[offset + col_idx] < cap_max_mat[offset + col_idx]) && !visited[col_idx]){
		new_frontier[col_idx] = true;
		par_mat[offset2 + row_idx] = true;
	}

	if(adj_mat[offset2 + row_idx] && (cap_mat[offset2 + row_idx] > 0) && !visited[col_idx]){
		new_frontier[col_idx] = true;
		par_mat[offset2 + row_idx] = true;
	}
}
__global__ void k2(const int N, bool* visited, int* frontier, bool* new_frontier){
	int cn = 0;
	for(int i=0;i<N;i++){
		if(new_frontier[i]){
			new_frontier[i] = false;
			frontier[++cn] = i;
			visited[i] = true;
		}
	}
	frontier[0] = cn;
}

std::vector<std::string> split(std::string str,std::string sep){
    char* cstr=const_cast<char*>(str.c_str());
    char* current;
    std::vector<std::string> arr;
    current=strtok(cstr,sep.c_str());
    while(current!=NULL){
        arr.push_back(current);
        current=strtok(NULL,sep.c_str());
    }
    return arr;
}

int main(int arg, char** argv){
	if(arg!=2){
		cout<<"usage: ./<out> <graph_size> <mygraph.txt"<<endl;
		return -1;
	}
	const int N = atoi(argv[1]);

	//read graph from <input>.txt
	bool* h_adj_mat = (bool*)malloc(N*N*sizeof(bool));
	int* h_cap_mat = (int*)malloc(N*N*sizeof(int));
	int* h_cap_max_mat = (int*)malloc(N*N*sizeof(int));

	for(int i=0;i<N*N;i++){
		string a;
		cin>>a;

		std::vector<std::string> arr;
    arr=split(a, ",");

		if(arr[0]=="1") h_adj_mat[i] = true;
		else h_adj_mat[i] = false;

		h_cap_mat[i] = atoi(arr[i].c_str());
		h_cap_max_mat[i] = atoi(arr[i].c_str());
	}

	bool* h_par_mat = (bool*)malloc(N*N*sizeof(bool));
	for(int i=0;i<N*N;i++)
		h_par_mat[i] = false;

	//generate visited and frontier vector; init them with node 0;
	bool* h_visited = (bool*)malloc(N*sizeof(bool));
	for(int i=0;i<N;i++) h_visited[i] = false;
	int* h_frontier = (int*)malloc(N*sizeof(int));
	bool* h_new_frontier = (bool*)malloc(N*sizeof(bool));
	for(int i=0;i<N;i++) h_new_frontier[i] = false;

	h_visited[0] = true;
	h_frontier[0] = 1;
	h_frontier[1] = 0;

	//malloc mem in gpu
	clock_t start,end, s, e;
	start = clock();
	bool *d_adj_mat, *d_par_mat, *d_visited, *d_new_frontier;
	int *d_cap_mat, *d_cap_max_mat, *d_frontier;
	hipMalloc((void**) &d_adj_mat, sizeof(bool) * N * N);
	hipMemcpy((void*) d_adj_mat, (void*) h_adj_mat, sizeof(bool)*N*N, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_cap_mat, sizeof(int) * N * N);
	hipMemcpy((void*) d_cap_mat, (void*) h_cap_mat, sizeof(int)*N*N, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_cap_max_mat, sizeof(int) * N * N);
	hipMemcpy((void*) d_cap_max_mat, (void*) h_cap_max_mat, sizeof(int)*N*N, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_par_mat, sizeof(bool) * N * N);
	hipMemcpy((void*) d_par_mat, (void*) h_par_mat, sizeof(bool)*N*N, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_visited, sizeof(bool) * N);
	hipMemcpy((void*) d_visited, (void*) h_visited, sizeof(bool)*N, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_frontier, sizeof(int) * (N+1));
	hipMemcpy((void*) d_frontier, (void*) h_frontier, sizeof(int)*N, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_new_frontier, sizeof(bool) * N);
	hipMemcpy((void*) d_new_frontier, (void*) h_new_frontier, sizeof(bool)*N, hipMemcpyHostToDevice);

	//loop until frontier vector is empty
	int cn =1;
	double t=0;
	while(h_frontier[0]!=0){
		cn+=h_frontier[0];
		//TODO:function call to update capacities in h_cap_mat here
		hipMemcpy((void*) d_cap_mat, (void*) h_cap_mat, sizeof(int)*N*N, hipMemcpyHostToDevice);

		//lauch kernel : launch threads to update frontier_len, visited and frontier in gpu local mem
		s=clock();
		kernel<<<h_frontier[0], N>>>(d_adj_mat,N,d_visited,d_frontier, d_new_frontier, d_par_mat, d_cap_mat, d_cap_max_mat);

		k2<<<1,1>>>(N, d_visited,d_frontier, d_new_frontier);
		e=clock();
		t+=double(e-s);

		hipMemcpy((void*) h_frontier, (void*) d_frontier, sizeof(int)*1, hipMemcpyDeviceToHost);
	}
	hipMemcpy((void*) h_par_mat, (void*) d_par_mat, sizeof(bool) * N * N, hipMemcpyDeviceToHost);

	end = clock();
	cout<<"queue through put: "<< cn<<endl;
	cout << "parallel BFS uses " << double(end - start) << " us in total"<< endl;
	cout << "kernel launching and computing uses " <<t<<" us"<<endl;
	cout << "mem copy uses " <<double(end - start) - t<<" us"<<endl;

	return 0;
}
