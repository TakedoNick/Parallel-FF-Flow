#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <time.h>

using namespace std;

//Check for edges valid to be part of augmented path
__global__ void kernel(bool* adj_mat, const int N, bool* visited, int* frontier, bool* new_frontier, bool* par_mat, int* cap_mat, int* cap_max_mat) {
	int row_idx = frontier[blockIdx.x+1];
	long offset = N * row_idx;

	int col_idx = threadIdx.x;
	long offset2 = N * col_idx;
	if(adj_mat[offset + col_idx] && (cap_mat[offset + col_idx] < cap_max_mat[offset + col_idx]) && !visited[col_idx]) {
		new_frontier[col_idx] = true;
		par_mat[offset2 + row_idx] = true;
	}

	if(adj_mat[offset2 + row_idx] && (cap_mat[offset2 + row_idx] > 0) && !visited[col_idx]) {
		new_frontier[col_idx] = true;
		par_mat[offset2 + row_idx] = true;
	}
}

//Update frontier
__global__ void k2(const int N, bool* visited, int* frontier, bool* new_frontier) {
	int count = 0;
	for(int i=0;i<N;i++) {
		if(new_frontier[i]) {
			new_frontier[i] = false;
			frontier[++count] = i;
			visited[i] = true;
		}
	}
	frontier[0] = count;
}

std::vector<std::string> split(std::string str,std::string sep) {
    char* cstr=const_cast<char*>(str.c_str());
    char* current;
    std::vector<std::string> arr;
    current=strtok(cstr,sep.c_str());
    while(current!=NULL) {
        arr.push_back(current);
        current=strtok(NULL,sep.c_str());
    }
    return arr;
}

int main(int arg, char** argv) {
	if(arg!=2) {
		cout<<"Please run in the following manner: ./<out> <graph_size> <<input>.txt"<<endl;
		return -1;
	}
	const int N = atoi(argv[1]);

	int maxflow = 0;

	//Read graph from <input>.txt
	bool* h_adj_mat = (bool*)malloc(N*N*sizeof(bool));
	int* h_cap_mat = (int*)malloc(N*N*sizeof(int));
	int* h_cap_max_mat = (int*)malloc(N*N*sizeof(int));

	for(int i=0;i<N*N;i++) {
		string a;
		cin>>a;

		std::vector<std::string> arr;
    arr=split(a, ",");

		if(arr[0]=="1") h_adj_mat[i] = true;
		else h_adj_mat[i] = false;

		h_cap_mat[i] = 0;
		h_cap_max_mat[i] = atoi(arr[1].c_str());

		//cout<<a<<":"<<h_adj_mat[i]<<" "<<h_cap_max_mat[i]<<"<>";
	}

	clock_t start, end, s, e;
	start = clock();

	//Allocate device memory for adj_mat, cap_mat and cap_max_mat
	bool *d_adj_mat, *d_par_mat, *d_visited, *d_new_frontier;
	int *d_cap_mat, *d_cap_max_mat, *d_frontier;
	hipMalloc((void**) &d_adj_mat, sizeof(bool) * N * N);
	hipMemcpy((void*) d_adj_mat, (void*) h_adj_mat, sizeof(bool)*N*N, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_cap_mat, sizeof(int) * N * N);
	hipMemcpy((void*) d_cap_mat, (void*) h_cap_mat, sizeof(int)*N*N, hipMemcpyHostToDevice);

	hipMalloc((void**) &d_cap_max_mat, sizeof(int) * N * N);
	hipMemcpy((void*) d_cap_max_mat, (void*) h_cap_max_mat, sizeof(int)*N*N, hipMemcpyHostToDevice);

	//Allocate host memory visited and frontier
	bool* h_par_mat = (bool*)malloc(N*N*sizeof(bool));
	bool* h_visited = (bool*)malloc(N*sizeof(bool));
	int* h_frontier = (int*)malloc(N*sizeof(int));
	bool* h_new_frontier = (bool*)malloc(N*sizeof(bool));

	//Allocate device memory for par_mat, visited and frontier
	hipMalloc((void**) &d_par_mat, sizeof(bool) * N * N);
	hipMalloc((void**) &d_visited, sizeof(bool) * N);
	hipMalloc((void**) &d_frontier, sizeof(int) * (N + 1));
	hipMalloc((void**) &d_new_frontier, sizeof(bool) * N);

	int* augPath = (int*)malloc(N*sizeof(int));

	double t = 0;
	while(1) {
		//Initialize par_mat
		for(int i=0;i<N*N;i++)
			h_par_mat[i] = false;

		//Initialize visited and frontier
		for(int i=0;i<N;i++) h_visited[i] = false;
		for(int i=0;i<N;i++) h_new_frontier[i] = false;

		h_visited[0] = true;
		h_frontier[0] = 1;
		h_frontier[1] = 0;

		//Copy to device memory for par_mat, visited and frontier
		hipMemcpy((void*) d_par_mat, (void*) h_par_mat, sizeof(bool)*N*N, hipMemcpyHostToDevice);
		hipMemcpy((void*) d_visited, (void*) h_visited, sizeof(bool)*N, hipMemcpyHostToDevice);
		hipMemcpy((void*) d_frontier, (void*) h_frontier, sizeof(int)*N, hipMemcpyHostToDevice);
		hipMemcpy((void*) d_new_frontier, (void*) h_new_frontier, sizeof(bool)*N, hipMemcpyHostToDevice);

		bool augFound = false;
		cout<<"\nFrontier:"<<endl<<endl;
		while(h_frontier[0] != 0) {
			for(int i = 0; i < h_frontier[0]; i++)
				cout<<h_frontier[i + 1]<<" ";
			cout<<endl;

			//Complete search if sink has been reached
			for(int i = 0; i < h_frontier[0]; i++)
				if(h_frontier[i + 1] == (N - 1)) {
					augFound = true;
					break;
				}

			//Call to kernels
			s = clock();
			kernel<<<h_frontier[0], N>>>(d_adj_mat,N,d_visited,d_frontier, d_new_frontier, d_par_mat, d_cap_mat, d_cap_max_mat);
			k2<<<1, 1>>>(N, d_visited, d_frontier, d_new_frontier);
			e = clock();
			t += double(e - s);

			hipMemcpy((void*) h_frontier, (void*) d_frontier, sizeof(int) * (N+1), hipMemcpyDeviceToHost);
		}

		if(augFound) {
			cout<<"\nAugmented path found!"<<endl;
			hipMemcpy((void*) h_par_mat, (void*) d_par_mat, sizeof(bool) * N * N, hipMemcpyDeviceToHost);

			cout<<"\nParent matrix:"<<endl<<endl;
			for(int i = 0; i < N; i++) {
				for(int j = 0; j < N; j++) {
					cout<<h_par_mat[i * N + j]<<" ";
				}
				cout<<endl;
			}

			//Find the augmented path
			augPath[0] = N - 1;
			int i = 1, vertex = N - 1;
			while(vertex != 0) {
				for(int j = 0; j < N; j++) {
					if(h_par_mat[vertex * N + j]) {
						vertex = j;
						augPath[i] = vertex;
						i++;
						break;
					}
				}
			}

			//Display augmented path
			cout<<"\nAugmented Path:\n\n";
			for(int i = 0; i < N; i++) {
				if(augPath[i] == 0) {
					cout<<augPath[i]<<endl;
					break;
				} else {
					cout<<augPath[i]<<" <- ";
				}
			}

			//Compute the bottleneck for the augmented path
			int bottleneck = -1;
			for(int i = 0; i < N; i++) {
				if(augPath[i] == 0)
					break;
				else {
					int k = augPath[i];
					int j = augPath[i + 1];
					int freeCap;
					if(h_adj_mat[j * N + k]) {
						freeCap = h_cap_max_mat[j * N + k] - h_cap_mat[j * N + k];
					} else {
						freeCap = h_cap_mat[k * N + j];
					}

					if(bottleneck == -1)
						bottleneck = freeCap;
					else if(freeCap < bottleneck)
						bottleneck = freeCap;
				}
			}
			cout<<"\nBottleneck of augmented path: "<<bottleneck<<endl;
			maxflow += bottleneck;

			//Update capacities in h_cap_mat
			for(int i = 0; i < N; i++) {
				if(augPath[i] == 0)
					break;
				else {
					int k = augPath[i];
					int j = augPath[i + 1];
					if(h_adj_mat[j * N + k]) {
						h_cap_mat[j * N + k] += bottleneck;
					} else {
						h_cap_mat[k * N + j] -= bottleneck;
					}
				}
			}

			hipMemcpy((void*) d_cap_mat, (void*) h_cap_mat, sizeof(int)*N*N, hipMemcpyHostToDevice);
		} else {
			cout<<"\nFord Fulkerson complete!\n";
			cout<<"\nMaximum Flow: "<<maxflow<<"\n";
			break;
		}
	}

	//Display execution times
	end = clock();
	cout<<"\nTime taken to run complete parallel Ford Fulkerson algorithm: "<<double(end - start)<<"us"<<endl;
	cout<<"Time taken to run kernel: "<<t<<"us"<<endl;
	cout<<"Time taken for memcpy from host to device: "<<double(end - start) - t<<"us"<<endl;

	//Free all memory
	free(augPath);
	free(h_par_mat);
	free(h_visited);
	free(h_frontier);
	free(h_new_frontier);
	free(h_adj_mat);
	free(h_cap_mat);
	free(h_cap_max_mat);
	hipFree(d_par_mat);
	hipFree(d_visited);
	hipFree(d_frontier);
	hipFree(d_new_frontier);
	hipFree(d_adj_mat);
	hipFree(d_cap_mat);
	hipFree(d_cap_max_mat);

	return 0;
}
